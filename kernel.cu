﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>

using namespace std;

// Function to check for CUDA errors and print error messages
void checkCudaError(hipError_t err, const char* msg) {
	if (err != hipSuccess) {
		cerr << "Error: " << msg << " - " << hipGetErrorString(err) << endl;
		exit(EXIT_FAILURE);
	}
}

int main() {
	int deviceCount; // Variable to store the number of CUDA devices
	hipError_t err; // Variable to store CUDA error codes

	// Get the number of CUDA devices available
	err = hipGetDeviceCount(&deviceCount);
	// Check for errors
	checkCudaError(err, "Failed to get device count");

	// If no CUDA devices are available, print an error message
	if (deviceCount == 0) {
		cout << "No CUDA devices found." << endl;
		return 0;
	}

	// Print the total number of CUDA devices
	cout << "Number of CUDA devices: " << deviceCount << "\n" << endl;
	

	// Loop through each CUDA device and print its properties
	for (int device = 0; device < deviceCount; device++) {
		hipDeviceProp_t deviceProp; // Variable to store device properties

		// Get the properties of the current CUDA device
		err = hipGetDeviceProperties(&deviceProp, device);
		// Check for errors
		checkCudaError(err, "Failed to get device properties");

		// Print the properties of the current CUDA device
		cout << "Device " << device+1 << ": " << deviceProp.name << endl;
		cout << " Compute Capability: " << deviceProp.major << ", " << deviceProp.minor << endl;
		cout << " Multiprocessors: " << deviceProp.multiProcessorCount << endl;
		cout << " Max Threads per Multiprocessor: " << deviceProp.maxThreadsPerMultiProcessor << endl;
		cout << " Max Threads per Block: " << deviceProp.maxThreadsPerBlock << endl;
		cout << " Clock Rate: " << deviceProp.clockRate << " kHz" << endl;
		cout << " Memory Clock Rate: " << deviceProp.memoryClockRate << " MHz" << endl;
		cout << " Memory Bus Width: " << deviceProp.memoryBusWidth << " bits" << endl;
		cout << " Memory Bandwidth: " << 2.0 * deviceProp.memoryBusWidth * deviceProp.memoryClockRate / 1e6 << " GB/s" << endl;
		cout << "\n" << endl;
	}

	return 0;
}
